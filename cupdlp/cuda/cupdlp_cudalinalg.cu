#include "cupdlp_cudalinalg.cuh"

extern "C" cupdlp_int cuda_alloc_MVbuffer(
    hipsparseHandle_t handle, hipsparseSpMatDescr_t cuda_csc,
    hipsparseDnVecDescr_t vecX, hipsparseDnVecDescr_t vecAx,
    hipsparseSpMatDescr_t cuda_csr, hipsparseDnVecDescr_t vecY,
    hipsparseDnVecDescr_t vecATy, void **dBuffer) {
  hipDataType computeType = HIP_R_32F;
#ifndef SFLOAT
  computeType = HIP_R_64F;
#endif

  size_t AxBufferSize = 0;
  size_t ATyBufferSize = 0;
  cupdlp_float alpha = 1.0;
  cupdlp_float beta = 0.0;
  // hipsparseSpSVAlg_t alg = HIPSPARSE_SPSV_ALG_DEFAULT;
  hipsparseSpMVAlg_t alg = HIPSPARSE_SPMV_CSR_ALG2; //deterministic

  // get the buffer size needed by csr Ax
  CHECK_CUSPARSE(hipsparseSpMV_bufferSize(
      handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &alpha, cuda_csr, vecX, &beta,
      vecAx, computeType, alg, &AxBufferSize))

  // get the buffer size needed by csc ATy
  CHECK_CUSPARSE(hipsparseSpMV_bufferSize(
      handle, HIPSPARSE_OPERATION_TRANSPOSE, &alpha, cuda_csc, vecY, &beta,
      vecATy, computeType, alg, &ATyBufferSize))

  size_t bufferSize =
      (AxBufferSize > ATyBufferSize) ? AxBufferSize : ATyBufferSize;

  // allocate an external buffer if needed
  CHECK_CUDA(hipMalloc(dBuffer, bufferSize))

  return EXIT_SUCCESS;
}

extern "C" cupdlp_int cuda_csc_Ax(hipsparseHandle_t handle,
                                  hipsparseSpMatDescr_t cuda_csc,
                                  hipsparseDnVecDescr_t vecX,
                                  hipsparseDnVecDescr_t vecAx, void *dBuffer,
                                  const cupdlp_float alpha,
                                  const cupdlp_float beta) {
  // hAx = alpha * Acsc * hX + beta * hAx

  hipsparseOperation_t op = HIPSPARSE_OPERATION_NON_TRANSPOSE;
  hipDataType computeType = HIP_R_32F;
#ifndef SFLOAT
  computeType = HIP_R_64F;
#endif

  CHECK_CUSPARSE(hipsparseSpMV(handle, op, &alpha, cuda_csc, vecX, &beta, vecAx,
                              // computeType, HIPSPARSE_SPMV_ALG_DEFAULT, dBuffer))
                              computeType, HIPSPARSE_SPMV_CSR_ALG2, dBuffer))

  return EXIT_SUCCESS;
}

extern "C" cupdlp_int cuda_csr_Ax(hipsparseHandle_t handle,
                                  hipsparseSpMatDescr_t cuda_csr,
                                  hipsparseDnVecDescr_t vecX,
                                  hipsparseDnVecDescr_t vecAx, void *dBuffer,
                                  const cupdlp_float alpha,
                                  const cupdlp_float beta) {
  // hAx = alpha * Acsc * hX + beta * hAx

  hipsparseOperation_t op = HIPSPARSE_OPERATION_NON_TRANSPOSE;
  hipDataType computeType = HIP_R_32F;
#ifndef SFLOAT
  computeType = HIP_R_64F;
#endif

  CHECK_CUSPARSE(hipsparseSpMV(handle, op, &alpha, cuda_csr, vecX, &beta, vecAx,
                              // computeType, HIPSPARSE_SPMV_ALG_DEFAULT, dBuffer))
                              computeType, HIPSPARSE_SPMV_CSR_ALG2, dBuffer))

  return EXIT_SUCCESS;
}

extern "C" cupdlp_int cuda_csc_ATy(hipsparseHandle_t handle,
                                   hipsparseSpMatDescr_t cuda_csc,
                                   hipsparseDnVecDescr_t vecY,
                                   hipsparseDnVecDescr_t vecATy, void *dBuffer,
                                   const cupdlp_float alpha,
                                   const cupdlp_float beta) {
  // hATy = alpha * Acsr^T * hY + beta * hATy
  hipsparseOperation_t op = HIPSPARSE_OPERATION_TRANSPOSE;
  hipDataType computeType = HIP_R_32F;
#ifndef SFLOAT
  computeType = HIP_R_64F;
#endif

  CHECK_CUSPARSE(hipsparseSpMV(handle, op, &alpha, cuda_csc, vecY, &beta, vecATy,
                              // computeType, HIPSPARSE_SPMV_ALG_DEFAULT, dBuffer))
                              computeType, HIPSPARSE_SPMV_CSR_ALG2, dBuffer))

  return EXIT_SUCCESS;
}

extern "C" cupdlp_int cuda_csr_ATy(hipsparseHandle_t handle,
                                   hipsparseSpMatDescr_t cuda_csr,
                                   hipsparseDnVecDescr_t vecY,
                                   hipsparseDnVecDescr_t vecATy, void *dBuffer,
                                   const cupdlp_float alpha,
                                   const cupdlp_float beta) {
  // hATy = alpha * Acsr^T * hY + beta * hATy
  hipsparseOperation_t op = HIPSPARSE_OPERATION_TRANSPOSE;
  hipDataType computeType = HIP_R_32F;
#ifndef SFLOAT
  computeType = HIP_R_64F;
#endif

  CHECK_CUSPARSE(hipsparseSpMV(handle, op, &alpha, cuda_csr, vecY, &beta, vecATy,
                              // computeType, HIPSPARSE_SPMV_ALG_DEFAULT, dBuffer))
                              computeType, HIPSPARSE_SPMV_CSR_ALG2, dBuffer))

  return EXIT_SUCCESS;
}

extern "C" void cupdlp_projSameub_cuda(cupdlp_float *x, const cupdlp_float ub,
                                       const cupdlp_int len) {
  element_wise_projSameub_kernel<<<cuda_gridsize(len), CUPDLP_BLOCK_SIZE>>>(
      x, ub, len);
}

extern "C" void cupdlp_projSamelb_cuda(cupdlp_float *x, const cupdlp_float lb,
                                       const cupdlp_int len) {
  element_wise_projSamelb_kernel<<<cuda_gridsize(len), CUPDLP_BLOCK_SIZE>>>(
      x, lb, len);
}

extern "C" void cupdlp_projub_cuda(cupdlp_float *x, const cupdlp_float *ub,
                                   const cupdlp_int len) {
  element_wise_projub_kernel<<<cuda_gridsize(len), CUPDLP_BLOCK_SIZE>>>(x, ub,
                                                                        len);
}

extern "C" void cupdlp_projlb_cuda(cupdlp_float *x, const cupdlp_float *lb,
                                   const cupdlp_int len) {
  element_wise_projlb_kernel<<<cuda_gridsize(len), CUPDLP_BLOCK_SIZE>>>(x, lb,
                                                                        len);
}

extern "C" void cupdlp_ediv_cuda(cupdlp_float *x, const cupdlp_float *y,
                                 const cupdlp_int len) {
  element_wise_div_kernel<<<cuda_gridsize(len), CUPDLP_BLOCK_SIZE>>>(x, y, len);
}

extern "C" void cupdlp_edot_cuda(cupdlp_float *x, const cupdlp_float *y,
                                 const cupdlp_int len) {
  element_wise_dot_kernel<<<cuda_gridsize(len), CUPDLP_BLOCK_SIZE>>>(x, y, len);
}

extern "C" void cupdlp_haslb_cuda(cupdlp_float *haslb, const cupdlp_float *lb,
                                  const cupdlp_float bound,
                                  const cupdlp_int len) {
  element_wise_initHaslb_kernal<<<cuda_gridsize(len), CUPDLP_BLOCK_SIZE>>>(
      haslb, lb, bound, len);
}

extern "C" void cupdlp_hasub_cuda(cupdlp_float *hasub, const cupdlp_float *ub,
                                  const cupdlp_float bound,
                                  const cupdlp_int len) {
  element_wise_initHasub_kernal<<<cuda_gridsize(len), CUPDLP_BLOCK_SIZE>>>(
      hasub, ub, bound, len);
}

extern "C" void cupdlp_filterlb_cuda(cupdlp_float *x, const cupdlp_float *lb,
                                     const cupdlp_float bound,
                                     const cupdlp_int len) {
  element_wise_filterlb_kernal<<<cuda_gridsize(len), CUPDLP_BLOCK_SIZE>>>(
      x, lb, bound, len);
}

extern "C" void cupdlp_filterub_cuda(cupdlp_float *x, const cupdlp_float *ub,
                                     const cupdlp_float bound,
                                     const cupdlp_int len) {
  element_wise_filterub_kernal<<<cuda_gridsize(len), CUPDLP_BLOCK_SIZE>>>(
      x, ub, bound, len);
}

extern "C" void cupdlp_initvec_cuda(cupdlp_float *x, const cupdlp_float val,
                                    const cupdlp_int len) {
  init_cuda_vec_kernal<<<cuda_gridsize(len), CUPDLP_BLOCK_SIZE>>>(x, val, len);
}

extern "C" void cupdlp_pgrad_cuda(cupdlp_float *xUpdate,
                                        const cupdlp_float *x,
                                        const cupdlp_float *cost,
                                        const cupdlp_float *ATy,
                                        const cupdlp_float dPrimalStep,
                                        const cupdlp_int len) {
    primal_grad_step_kernal<<<cuda_gridsize(len), CUPDLP_BLOCK_SIZE>>>(
        xUpdate, x, cost, ATy, dPrimalStep, len);
}

extern "C" void cupdlp_dgrad_cuda(cupdlp_float *yUpdate, const cupdlp_float *y, const cupdlp_float *b,
    const cupdlp_float *Ax, const cupdlp_float *AxUpdate,
    const cupdlp_float dDualStep, const cupdlp_int len) {
      dual_grad_step_kernal<<<cuda_gridsize(len), CUPDLP_BLOCK_SIZE>>>(
          yUpdate, y, b, Ax, AxUpdate, dDualStep, len);
}

extern "C" void cupdlp_sub_cuda(cupdlp_float *z, const cupdlp_float *x,
                                  const cupdlp_float *y, const cupdlp_int len)
{
   naive_sub_kernal<<<cuda_gridsize(len), CUPDLP_BLOCK_SIZE>>>(z, x, y, len);
}