#include "hip/hip_runtime.h"
#include "cupdlp_cuda_kernels.cuh"

dim3 cuda_gridsize(cupdlp_int n) {
  cupdlp_int k = (n - 1) / CUPDLP_BLOCK_SIZE + 1;
  cupdlp_int x = k;
  cupdlp_int y = 1;
  if (x > 65535) {
    x = ceil(sqrt(k));
    y = (n - 1) / (x * CUPDLP_BLOCK_SIZE) + 1;
  }
  dim3 d = {(unsigned int)x, (unsigned int)y, 1};
  return d;
}

__global__ void element_wise_dot_kernel(cupdlp_float *x, const cupdlp_float *y,
                                        const cupdlp_int len) {
  cupdlp_int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < len) x[i] *= y[i];
}

__global__ void element_wise_div_kernel(cupdlp_float *x, const cupdlp_float *y,
                                        const cupdlp_int len) {
  cupdlp_int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < len) x[i] /= y[i];
}

__global__ void element_wise_projlb_kernel(cupdlp_float *x,
                                           const cupdlp_float *lb,
                                           const cupdlp_int len) {
  cupdlp_int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < len) x[i] = x[i] < lb[i] ? lb[i] : x[i];
}

__global__ void element_wise_projub_kernel(cupdlp_float *x,
                                           const cupdlp_float *ub,
                                           const cupdlp_int len) {
  cupdlp_int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < len) x[i] = x[i] > ub[i] ? ub[i] : x[i];
}

__global__ void element_wise_projSamelb_kernel(cupdlp_float *x,
                                               const cupdlp_float lb,
                                               const cupdlp_int len) {
  cupdlp_int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < len) x[i] = x[i] <= lb ? lb : x[i];
}

__global__ void element_wise_projSameub_kernel(cupdlp_float *x,
                                               const cupdlp_float ub,
                                               const cupdlp_int len) {
  cupdlp_int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < len) x[i] = x[i] >= ub ? ub : x[i];
}

__global__ void element_wise_initHaslb_kernel(cupdlp_float *haslb,
                                              const cupdlp_float *lb,
                                              const cupdlp_float bound,
                                              const cupdlp_int len) {
  cupdlp_int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < len) haslb[i] = lb[i] > bound ? 1.0 : 0.0;
}

__global__ void element_wise_initHasub_kernel(cupdlp_float *hasub,
                                              const cupdlp_float *ub,
                                              const cupdlp_float bound,
                                              const cupdlp_int len) {
  cupdlp_int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < len) hasub[i] = ub[i] < bound ? 1.0 : 0.0;
}

__global__ void element_wise_filterlb_kernel(cupdlp_float *x,
                                             const cupdlp_float *lb,
                                             const cupdlp_float bound,
                                             const cupdlp_int len) {
  cupdlp_int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < len) x[i] = lb[i] > bound ? lb[i] : 0.0;
}

__global__ void element_wise_filterub_kernel(cupdlp_float *x,
                                             const cupdlp_float *ub,
                                             const cupdlp_float bound,
                                             const cupdlp_int len) {
  cupdlp_int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < len) x[i] = ub[i] < bound ? ub[i] : 0.0;
}

__global__ void init_cuda_vec_kernel(cupdlp_float *x, const cupdlp_float val,
                                     const cupdlp_int len) {
  cupdlp_int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < len) x[i] = val;
}

// xUpdate = proj(x - dPrimalStep * (cost - ATy))
__global__ void primal_grad_step_kernel(cupdlp_float *__restrict__ xUpdate,
                                        const cupdlp_float * __restrict__ x,
                                        const cupdlp_float * __restrict__ cost,
                                        const cupdlp_float * __restrict__ ATy,
                                        const cupdlp_float * __restrict__ lb,
                                        const cupdlp_float * __restrict__ ub,
                                        cupdlp_float dPrimalStep, int nCols) {
  for (int i = blockDim.x * blockIdx.x + threadIdx.x; i < nCols; i += gridDim.x * blockDim.x) {
    xUpdate[i] = min(max(cupdlp_fma_rn(dPrimalStep, ATy[i] - cost[i], x[i]), lb[i]), ub[i]);
  }
}

// yUpdate = proj(y + dDualStep * (b - 2 AxUpdate + Ax))
__global__ void dual_grad_step_kernel(cupdlp_float * __restrict__ yUpdate,
                                      const cupdlp_float * __restrict__ y,
                                      const cupdlp_float * __restrict__ b,
                                      const cupdlp_float * __restrict__ Ax,
                                      const cupdlp_float * __restrict__ AxUpdate,
                                      cupdlp_float dDualStep, int nRows, int nEqs) {
  for (int i = blockDim.x * blockIdx.x + threadIdx.x; i < nRows; i += gridDim.x * blockDim.x) {
    cupdlp_float upd = cupdlp_fma_rn(dDualStep, b[i] - 2 * AxUpdate[i] + Ax[i], y[i]);
    yUpdate[i] = i >= nEqs ? max(upd, 0.0) : upd;
  }
}

// z = x - y
__global__ void naive_sub_kernel(cupdlp_float *z, const cupdlp_float *x,
                                  const cupdlp_float *y, const cupdlp_int len) {
  cupdlp_int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < len) z[i] = x[i] - y[i];
}


#define QUARTER_WARP_REDUCE_2(val1, val2) { \
  val1 += __shfl_down_sync(0xFFFFFFFF, val1, 4); \
  val2 += __shfl_down_sync(0xFFFFFFFF, val2, 4); \
  val1 += __shfl_down_sync(0xFFFFFFFF, val1, 2); \
  val2 += __shfl_down_sync(0xFFFFFFFF, val2, 2); \
  val1 += __shfl_down_sync(0xFFFFFFFF, val1, 1); \
  val2 += __shfl_down_sync(0xFFFFFFFF, val2, 1); \
}

#define FULL_WARP_REDUCE_2(val1, val2) { \
  val1 += __shfl_down_sync(0xFFFFFFFF, val1, 16); \
  val2 += __shfl_down_sync(0xFFFFFFFF, val2, 16); \
  val1 += __shfl_down_sync(0xFFFFFFFF, val1, 8); \
  val2 += __shfl_down_sync(0xFFFFFFFF, val2, 8); \
  val1 += __shfl_down_sync(0xFFFFFFFF, val1, 4); \
  val2 += __shfl_down_sync(0xFFFFFFFF, val2, 4); \
  val1 += __shfl_down_sync(0xFFFFFFFF, val1, 2); \
  val2 += __shfl_down_sync(0xFFFFFFFF, val2, 2); \
  val1 += __shfl_down_sync(0xFFFFFFFF, val1, 1); \
  val2 += __shfl_down_sync(0xFFFFFFFF, val2, 1); \
}

// assumes block size = 256, warp size = 32
__global__ void movement_1_kernel(cupdlp_float * __restrict__ res_x, cupdlp_float * __restrict__ res_y,
                                  const cupdlp_float * __restrict__ xUpdate, const cupdlp_float * __restrict__ x,
                                  const cupdlp_float * __restrict__ atyUpdate, const cupdlp_float * __restrict__ aty,
                                  int nCols) {

  __shared__ cupdlp_float shared_x[32];
  __shared__ cupdlp_float shared_y[32];
  cupdlp_float val_x = 0.0;
  cupdlp_float val_y = 0.0;
  for (int i = blockDim.x * blockIdx.x + threadIdx.x; i < nCols; i += blockDim.x * gridDim.x) {
      cupdlp_float dx = xUpdate[i] - x[i];
      cupdlp_float day = atyUpdate[i] - aty[i];
      val_x = cupdlp_fma_rn(dx, dx, val_x);
      val_y = cupdlp_fma_rn(day, dx, val_y);
  }

  int lane = threadIdx.x % 32;
  int wid = threadIdx.x / 32;

  FULL_WARP_REDUCE_2(val_x, val_y)
  if (lane == 0) {
    shared_x[wid] = val_x;
    shared_y[wid] = val_y;
  }
  __syncthreads();

  if (wid == 0) {
    val_x = (threadIdx.x < blockDim.x / 32) ? shared_x[lane] : 0.0;
    val_y = (threadIdx.x < blockDim.x / 32) ? shared_y[lane] : 0.0;
    QUARTER_WARP_REDUCE_2(val_x, val_y)
    if (threadIdx.x == 0) {
      res_x[blockIdx.x] = val_x;
      res_y[blockIdx.x] = val_y;
    }
  }
}

#define QUARTER_WARP_REDUCE(val) { \
  val += __shfl_down_sync(0xFFFFFFFF, val, 4); \
  val += __shfl_down_sync(0xFFFFFFFF, val, 2); \
  val += __shfl_down_sync(0xFFFFFFFF, val, 1); \
}

#define HALF_WARP_REDUCE(val) { \
  val += __shfl_down_sync(0xFFFFFFFF, val, 8); \
  val += __shfl_down_sync(0xFFFFFFFF, val, 4); \
  val += __shfl_down_sync(0xFFFFFFFF, val, 2); \
  val += __shfl_down_sync(0xFFFFFFFF, val, 1); \
}

#define FULL_WARP_REDUCE(val) { \
  val += __shfl_down_sync(0xFFFFFFFF, val, 16); \
  val += __shfl_down_sync(0xFFFFFFFF, val, 8); \
  val += __shfl_down_sync(0xFFFFFFFF, val, 4); \
  val += __shfl_down_sync(0xFFFFFFFF, val, 2); \
  val += __shfl_down_sync(0xFFFFFFFF, val, 1); \
}

// assumes: block size = 256, warp size = 32
__global__ void movement_2_kernel(cupdlp_float * __restrict__ res,
                                  const cupdlp_float * __restrict__ yUpdate, const cupdlp_float * __restrict__ y,
                                  int nRows) {

  __shared__ cupdlp_float shared[32];
  cupdlp_float val = 0.0;
  for (int i = blockDim.x * blockIdx.x + threadIdx.x; i < nRows; i += blockDim.x * gridDim.x) {
      cupdlp_float d = yUpdate[i] - y[i];
      val = cupdlp_fma_rn(d, d, val);
  }

  int lane = threadIdx.x % 32;
  int wid = threadIdx.x / 32;

  FULL_WARP_REDUCE(val)
  if (lane == 0) {
    shared[wid] = val;
  }
  __syncthreads();

  if (wid == 0) {
    val = (threadIdx.x < blockDim.x / 32) ? shared[lane] : 0.0;
    QUARTER_WARP_REDUCE(val)
    if (threadIdx.x == 0) {
      res[blockIdx.x] = val;
    }
  }
}

// assumes: block size = 512, warp size = 32
__global__ void sum_kernel(cupdlp_float * __restrict__ res, const cupdlp_float * __restrict__ x, int n) {

  __shared__ cupdlp_float shared[32];
  cupdlp_float val = 0.0;
  for (int i = blockDim.x * blockIdx.x + threadIdx.x; i < n; i += blockDim.x * gridDim.x) {
      val += x[i];
  }

  int lane = threadIdx.x % 32;
  int wid = threadIdx.x / 32;

  FULL_WARP_REDUCE(val)
  if (lane == 0) {
    shared[wid] = val;
  }
  __syncthreads();

  if (wid == 0) {
    val = (threadIdx.x < blockDim.x / 32) ? shared[lane] : 0.0;
    HALF_WARP_REDUCE(val)
    if (threadIdx.x == 0) {
      res[blockIdx.x] = val;
    }
  }
}
